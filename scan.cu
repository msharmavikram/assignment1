#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

/*	  
void GPUMalloc (float **d_A, int size){

   hipError_t err = hipMalloc((void **) d_A, size);

   if (err != hipSuccess){
      printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
   }
}*/	  
	  
__global__ void finalsum(float *output, float *input, int len) {
	//For first block dont compute. results are already final. 
	if(blockIdx.x == 0) return;
	
	unsigned int idx = 2* blockIdx.x * BLOCK_SIZE + threadIdx.x;
	
	if (idx < len) 
		output[idx] += input[blockIdx.x -1]; // final value 
	if ((idx + BLOCK_SIZE) < len)
		output[idx+BLOCK_SIZE] += input[blockIdx.x-1];
}
	
__global__ void scan(float *input, float *output, float *intermediate, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host

  //@@ This is Brent Kung scan kernel implementation. 
  __shared__ float sharedMem[BLOCK_SIZE*2]; 
  int idx = 2*blockIdx.x * BLOCK_SIZE + threadIdx.x; 
  //@@ get two set of values from the memory. i and i+block_size
  if( idx <len) 
	  sharedMem[threadIdx.x] = input[idx];
  else 
	  sharedMem[threadIdx.x] = 0.0;
  if( idx+ BLOCK_SIZE <len) 
	  sharedMem[threadIdx.x+ BLOCK_SIZE] = input[idx+ BLOCK_SIZE];
  else 
	  sharedMem[threadIdx.x+BLOCK_SIZE]  = 0.0;
	  
  __syncthreads(); //sync all inputs
  //forward path  
  for (unsigned int stride = 1; stride <= BLOCK_SIZE;stride *=2){
	int index = (threadIdx.x+1) * 2* stride -1; 
	if( index < 2*BLOCK_SIZE) 
		sharedMem[index] += sharedMem[index-stride]; 
        __syncthreads(); //sync all inputs
  }
  
  //backward path -- reduction
  for (unsigned int stride = BLOCK_SIZE/2; stride >0 ; stride /=2){
	int index = (threadIdx.x + 1)*stride*2 -1;
	if(index + stride < 2* BLOCK_SIZE)
	        sharedMem[index+stride] += sharedMem[index];
	__syncthreads();//wait until previous compute are over. 
  }
  
  if(idx <len) output[idx] = sharedMem[threadIdx.x];
  if(idx+ BLOCK_SIZE < len) output[idx+BLOCK_SIZE] = sharedMem[threadIdx.x + BLOCK_SIZE];
  
  //if its first kernel then do add data to intermediate stage 
  if(intermediate !=NULL && threadIdx.x ==0) 
	  intermediate[blockIdx.x] = sharedMem[2*BLOCK_SIZE-1];
  
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  float *devicek2Input;
  float *devicek2Output;
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ", numElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&devicek2Input, BLOCK_SIZE*2 * sizeof(float)));
  wbCheck(hipMalloc((void **)&devicek2Output, BLOCK_SIZE*2 * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid((numElements-1)/(BLOCK_SIZE*2)+1, 1, 1);// not doing optimization as in reduction.
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
 
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  //First kernel does Bernt kung scan
  scan<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, devicek2Input, numElements);
  hipDeviceSynchronize();
  //second kernel is same as scan but over aggregated sum of the first results. 
  // second kernel is one block only and it should write anything out.
  scan<<<dim3(1,1,1), dimBlock>>>(devicek2Input, devicek2Output, NULL, BLOCK_SIZE*2);
  hipDeviceSynchronize();
  //third kernel adder of each results. 
  // Half of the elements in the deviceOutput are done compute. 
  //size of this kernel will be equal to size of input kernel. 
  finalsum<<<dimGrid, dimBlock>>>(deviceOutput, devicek2Output, numElements); 
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  hipFree(devicek2Input);
  hipFree(devicek2Output);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}

